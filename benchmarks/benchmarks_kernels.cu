#include "hip/hip_runtime.h"
/****
 * Copyright (c) 2011-2014, NVIDIA Corporation.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 *    * Redistributions of source code must retain the above copyright notice,
 *      this list of conditions and the following disclaimer.
 *    * Redistributions in binary form must reproduce the above copyright
 *      notice, this list of conditions and the following disclaimer in the
 *      documentation and/or other materials provided with the distribution.
 *    * Neither the name of the NVIDIA Corporation nor the names of its
 *      contributors may be used to endorse or promote products derived from
 *      this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF
 * THE POSSIBILITY OF SUCH DAMAGE.
 ****/

#include "benchmarks_kernels.hpp"
static const int over_sub_factor = 2;

// ============ Working only with CUDA or LibGDSync  ============ 
__global__ void calc_kernel(int n, float c, float *in, float *out)
{
        const uint tid = threadIdx.x;
        const uint bid = blockIdx.x;
        const uint block_size = blockDim.x;
        const uint grid_size = gridDim.x;
        const uint gid = tid + bid*block_size;
        const uint n_threads = block_size*grid_size;
        for (int i=gid; i<n; i += n_threads)
                out[i] = in[i] * c;
}

int gpu_launch_calc_kernel(size_t size, int gpu_num_sm, hipStream_t stream)
{
        const int nblocks = over_sub_factor * gpu_num_sm;
        const int nthreads = 32*2;
        int n = size / sizeof(float);
        static float *in = NULL;
        static float *out = NULL;
        if (!in) {
            CUDA_CHECK(hipMalloc((void **)&in, size));
            CUDA_CHECK(hipMalloc((void **)&out, size));

            CUDA_CHECK(hipMemset((void *)in, 1, size));
            CUDA_CHECK(hipMemset((void *)out, 1, size));
        }
        calc_kernel<<<nblocks, nthreads, 0, stream>>>(n, 1.0f, in, out);
        CUDA_CHECK(hipGetLastError());
        return 0;
}

__global__ void dummy_kernel(double time, double clockrate)
{
    long long int start, stop;
    double usec;
    volatile int counter;

    start = clock64();
    do {
        stop = clock64();
    usec = ((double)(stop-start)*1000)/((double)clockrate); 
    counter = usec;
    } while(usec < time);
}

int gpu_launch_dummy_kernel(double time, double clockrate, hipStream_t stream)
{
    dummy_kernel <<<1, 1, 0, stream>>>(time, clockrate);
    CUDA_CHECK(hipGetLastError());
    return 0;
}
